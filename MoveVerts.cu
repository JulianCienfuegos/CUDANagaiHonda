#include <vector>
#include "coordinate.hpp"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>

void MoveVerts(std::vector<coordinate>& coord, std::vector<double> tx, std::vector<double> ty)
{
    // Get all of the x and y verts from the coordinate list
    // store them in two x and y vectors.
//std::cout <<"Going to the GPU!" << std::endl;
    std::vector<double> x;
    std::vector<double> y;
    for(std::vector<coordinate>::iterator c = coord.begin();
        c != coord.end(); c++)
    {   
        x.push_back(c->x);
        y.push_back(c->y);
    }
    
    // Get all of the data on the GPU
    thrust::device_vector<double> X  (x.begin(), x.end());
    thrust::device_vector<double> Y  (y.begin(), y.end());
    thrust::device_vector<double> TX (tx.begin(), tx.end());
    thrust::device_vector<double> TY (ty.begin(), ty.end());

    // Add TX to X and store it in X
    thrust::transform(X.begin(), X.end(), 
                     TX.begin(),
                     X.begin(), 
                     thrust::plus<double>());
    
    // Copy back to x
    thrust::copy(X.begin(), X.end(), x.begin());
    
    // Repeat for y.
    thrust::transform(Y.begin(), Y.end(), 
                     TY.begin(),
                     Y.begin(), 
                     thrust::plus<double>());
                     
    thrust::copy(Y.begin(), Y.end(), Y.begin());
    
    // Now put all of the data back into the coordinates.
    for(int i = 0; i < coord.size(); i++)
        coord[i].SetXY(x[i], y[i]);
//    std::cout << "Done on the GPU" << std::endl;
}

